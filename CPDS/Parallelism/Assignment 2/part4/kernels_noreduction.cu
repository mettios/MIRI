#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__global__ void gpu_Heat (float *h, float *g, int N) {

	//kernel computation
	int j = threadIdx.x + blockIdx.x * blockDim.x; // column
	int i = threadIdx.y + blockIdx.y * blockDim.y; // row
	int index = i * N + j;
	// check corners
	if (i > 0 && i < (N-1) && j > 0 && j < (N-1)) {
		g[index] = 0.25 * (h[i * N + (j-1)] + // left
				h[i * N + (j+1)] + 	  // right
				h[(i-1) * N + j] +  	  // top
				h[(i+1) * N + j]);  	  // bottom
	}
}
