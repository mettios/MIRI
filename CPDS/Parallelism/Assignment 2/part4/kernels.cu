#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__global__ void gpu_Heat (float *h, float *g, int N, float* residuals) {
	// added residuals as an additional input for this function
	extern __shared__ float sdata[]; // shared memory for residuals
	int j = threadIdx.x + blockIdx.x * blockDim.x; // column
	int i = threadIdx.y + blockIdx.y * blockDim.y; // row
	int index = i * N + j; // index for jacobi computation
	unsigned int tid = threadIdx.x * blockDim.x + threadIdx.y; // unique id of the thread for this block
	// check corners
	if (i > 0 && i < (N-1) && j > 0 && j < (N-1)) {
		g[index] = 0.25 * (h[i * N + (j-1)] +     // left
				h[i * N + (j+1)] + 	  // right
				h[(i-1) * N + j] +  	  // top
				h[(i+1) * N + j]);  	  // bottom
	// Calculate diff value
		float diff = g[index] - h[index];
		sdata[tid] = diff * diff;
	}
	// wait for threads 
	__syncthreads();
	// apply reduction
	for (unsigned int s = blockDim.x * blockDim.y / 2; s > 32; s >>= 1) {
		if (tid > s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	// unroll last warp
	if (tid < 32) {
		sdata[tid] += sdata[tid + 32];
		sdata[tid] += sdata[tid + 16];
		sdata[tid] += sdata[tid + 8];
		sdata[tid] += sdata[tid + 4];
		sdata[tid] += sdata[tid + 2];
		sdata[tid] += sdata[tid + 1];
	}
	// Thread 0 holds the residual for this block
	if (tid == 0) {
		residuals[blockIdx.x * gridDim.y + blockIdx.y] = sdata[0];
	}
}
